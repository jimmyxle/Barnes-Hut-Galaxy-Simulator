#include "hip/hip_runtime.h"
#include "Galaxy.h"
#include <ctime>
#include "tbb/parallel_for.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <sstream>



__global__ void calcDistance_GPU(Vector2D* forces, Vector2D* particles, int count)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < count)
	{
		Vector2D& thisForce = forces[idx];
		Vector2D& thisParticle = particles[idx];

		for (int i = 0; i < count; i++)
		{
			if (i == idx) continue;


			//timestep per calculation
			double TIME = 0.0015;
			{
				//prevents NaN problems
				if (thisForce.x != thisForce.x)
					thisForce.x = 0;

				if (thisForce.y != thisForce.y)
					thisForce.y = 0;
			}

			double acc_x = forces[idx].x;
			double acc_y = forces[idx].y;
			//prevents points from accelerating too far from the center
			double max = 1.0 / 25;
			if (acc_x >= max)
			{
				acc_x = max;
			}
			if (acc_x < -max)
			{
				acc_x = -max;
			}

			if (acc_y >= max)
			{
				acc_y = max;
			}
			if (acc_y < -max)
			{
				acc_y = -max;
			}

			//velocities

			thisParticle.vx += acc_x * TIME;
			thisParticle.vy += acc_y * TIME;

			//positions
			thisParticle.x += thisParticle.vx;
			thisParticle.y += thisParticle.vy;

			//bounce particles off the borders
			if (thisParticle.x >= 0.99)
			{
				thisParticle.x = 0.99;
				thisParticle.vx *= -0.5;
			}
			if (thisParticle.x <= -0.99)
			{
				thisParticle.x = -0.99;
				thisParticle.vx *= -0.5;
			}
			if (thisParticle.y >= 0.99)
			{
				thisParticle.y = 0.99;
				thisParticle.vy *= -0.5;
			}
			if (thisParticle.y <= -0.99)
			{
				thisParticle.y = -0.99;
				thisParticle.vy *= -0.5;
			}


		}
	}


}



Galaxy::Galaxy()
{
	NUMBER_PARTICLES = 300;

	allParticles.reserve(NUMBER_PARTICLES);
	allParticles = particle.generateParticles(0, 0, NUMBER_PARTICLES, 0.5, 80000,0,0);
	
	//max is bot right corner , min is top left corner
	max = new Vector2D(1, -1, 0, 0);
	min = new Vector2D(-1, 1, 0, 0);
	root = new QuadNode(*min, *max, QuadNode::NONE, nullptr);
}

Galaxy::Galaxy(double _x, double _y, double _centerMass, int _NUM_P,
	double vel_x, double vel_y, double radius)
{
	NUMBER_PARTICLES = _NUM_P;

	allParticles.reserve(NUMBER_PARTICLES );
	//pass x,y which are the center of the galaxy
	//number of particles to make
	//radius of galaxy initially
	x = _x;
	y = _y;
	allParticles = particle.generateParticles(x, y, NUMBER_PARTICLES, radius,
		_centerMass, vel_x, vel_y);
	double boxSize = 3 ;
	max = new Vector2D(boxSize, -boxSize, 0, 0);
	min = new Vector2D(-boxSize, boxSize, 0, 0);

	root = new QuadNode(*min, *max, QuadNode::NONE, nullptr);
}

Galaxy::~Galaxy()
{
	delete root;
	delete max;
	delete min;
	for (auto it = allParticles.begin(); it != allParticles.end(); it++)
	{
		delete (*it);
	}
}

/*
	this adds ones planets to the other list and change num planets var in main 
*/
void Galaxy::add_galaxy(Galaxy& galaxy, double vel_x, double vel_y)
{
	for (auto it = galaxy.allParticles.begin(); it != galaxy.allParticles.end(); it++)
	{
		allParticles.push_back(*it);
	}
	NUMBER_PARTICLES *= 2;
}

//following functions are for drawing points
void Galaxy::displayParticles(std::vector<ParticleData*> arr) 
{
	glClearColor(0, 0, 0, 0);
	//clear color and depth buffer
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glColor3f(0, 255, 0);
	glPointSize(4);
	glBegin(GL_POINTS);

	int SIZE = arr.size();
	bool init = true;
	do
	{
		double x = (arr[0])->xy->x;
		double y = (arr[0])->xy->y;
		glVertex2d(x, y);
		init = !init;
	} while (init);

	glEnd();
	glPointSize(1);
	glBegin(GL_POINTS);
	glColor3f(1, 1, 1);

	for (int i = 1; i < SIZE; i++)
	{
		double x = (arr[i])->xy->x;
		double y = (arr[i])->xy->y;
		glVertex2d(x, y);
	}

	glEnd();
	glPopMatrix();
}

void Galaxy::displayParticles(std::vector<ParticleData*> arr1, std::vector<ParticleData*> arr2)
{
	glClearColor(0, 0, 0, 0);
	//clear color and depth buffer
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glPointSize(4);

	glBegin(GL_POINTS);

	int SIZE = arr1.size();

	bool init = true;
	do
	{
		glColor3f(255, 0, 0);

		double x = (arr1[0])->xy->x;
		double y = (arr1[0])->xy->y;
		glVertex2d(x, y);

		x = (arr1[SIZE / 2])->xy->x;
		y = (arr1[SIZE / 2])->xy->y;
		glVertex2d(x, y);


		init = !init;
	} while (init);

	glEnd();

	glPointSize(1);
	glBegin(GL_POINTS);

	glColor3f(0, 255, 0);
	for (int i = 1; i < SIZE/2; i++)
	{
		double x = (arr1[i])->xy->x;
		double y = (arr1[i])->xy->y;
		glVertex2d(x, y);
	}

	glColor3f(0, 255, 255);

	for (int i = SIZE/2+1 ; i < SIZE; i++)
	{
		double x = (arr1[i])->xy->x;
		double y = (arr1[i])->xy->y;
		glVertex2d(x, y);
	}
	glEnd();
	glPopMatrix();
}


void Galaxy::recursiveBoxes(QuadNode& qt, double factor)
{
	glBegin(GL_POLYGON);
	glVertex2d(factor*qt.getVector(0).x, factor*qt.getVector(0).y);
	glVertex2d(factor*qt.getVector(2).x, factor*qt.getVector(0).y);
	glVertex2d(factor*qt.getVector(2).x, factor*qt.getVector(2).y);
	glVertex2d(factor*qt.getVector(0).x, factor*qt.getVector(2).y);


	glEnd();

	for (std::vector<QuadNode*>::iterator it = (qt).nodeArr.begin();
		it != (qt).nodeArr.end(); it++)
	{
		recursiveBoxes(*(*it), 1.0);
	}
}


void Galaxy::displayQuadrant(QuadNode& quad)
{

	glColor3f(255, 0, 0);
	glPointSize(1);
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);



	recursiveBoxes(quad, 1.0);


	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

	glPopMatrix();
}
void Galaxy::displayQuadrant(QuadNode& quad, QuadNode& second)
{

	glColor3f(255, 0, 0);
	glPointSize(1);
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);



	recursiveBoxes(quad, 1.0);
	recursiveBoxes(second, 1.0);


	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

	glPopMatrix();
}

int Galaxy::running_display()
{
	std::cout << "running display mode" << std::endl;
	Vector2D target;

	GLFWwindow* window;
	if (!glfwInit())
	{
		std::cout << "Error Initializing GLFW" << std::endl;
		return -1;
	}
	window = glfwCreateWindow(750, 750, "Barnes-Hut Tree: Single Galaxy", NULL, NULL);
	if (!window)
	{
		std::cout << "Error creating window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);

	//std::vector<Vector2D> forces1(NUMBER_PARTICLES);

	size_t max = NUMBER_PARTICLES;
	Vector2D* forces1[ 150 ];

	for (unsigned int i = 0; i < max; i++)
	{
		forces1[i] = new Vector2D();
	}

	double deltaTime = 0;
	unsigned int frames = 0;
	double  frameRate = 30;
	double  averageFrameTimeMilliseconds = 33.333;

	std::clock_t end;
	std::clock_t begin;
	std::clock_t start;
	double time;

	while (!glfwWindowShouldClose(window))
	{
		begin = clock();
		start = begin;


		//task parallel
		root->buildTree(allParticles, NUMBER_PARTICLES);

		//data parallelism
		root->computeMassDistribution();

		//uncomment these to show particles/quadrants
		displayParticles(allParticles);
		//displayQuadrant(*root);

		/**/
		//calc forces 
		//data parallel

		tbb::parallel_for(size_t(0), max, [&](size_t i) {
			root->calcForce(*(allParticles[i]),  *(forces1[i]) );	
		});

		/*here we try to do cuda first*/

		//make array of forces
		//Vector2D *h_force = new Vector2D[max];
		Vector2D *d_force;

		Vector2D *h_particles = new Vector2D[max];
		Vector2D *d_particles;
		
		for (unsigned int i = 0; i < max; i++)
		{
			//h_force[i] = *forces1[i];
			h_particles[i] = *(allParticles[i])->xy;
		}
		
		if (hipMalloc(&d_force, sizeof(Vector2D)*max) != hipSuccess)
		{
			//delete[] d_force;
			std::stringstream ss;
			ss << "Can't malloc d_force_x.";
			throw std::runtime_error(ss.str());
		}
		if (hipMalloc(&d_particles, sizeof(Vector2D)*max) != hipSuccess)
		{
			delete[] d_force;
			std::stringstream ss;
			ss << "Can't malloc d_force_x.";
			throw std::runtime_error(ss.str());
		}

		if (hipMemcpy(d_force, forces1, sizeof(Vector2D)*max, hipMemcpyHostToDevice) != hipSuccess)
		{
			hipFree(d_force);
	//		delete[] forces1;
			hipFree(d_particles);
			delete[] h_particles;
			std::stringstream ss;
			ss << "Can't copy  host force_x.";
			throw std::runtime_error(ss.str());
		}

		if (hipMemcpy(d_particles, h_particles, sizeof(Vector2D)*max, hipMemcpyHostToDevice) != hipSuccess)
		{
			hipFree(d_force);
	//		delete[] forces1;
			hipFree(d_particles);
			delete[] h_particles;
			std::stringstream ss;
			ss << "Can't copy  host force_x.";
			throw std::runtime_error(ss.str());
		}

		 //kernel 
	//kernal <<< 1231,13123,1321 >>();
		calcDistance_GPU<<<((max / 32) + 1), 32>>>(d_force, d_particles, max);
		//hipDeviceSynchronize();



		if (hipMemcpy(h_particles , d_particles, sizeof(Vector2D)*max, hipMemcpyDeviceToHost) != hipSuccess)
		{
			hipFree(d_force);
			//delete[] forces1;
			hipFree(d_particles);
			delete[] h_particles;
			std::stringstream ss;
			ss << "Can't copy device d_force_x.";
			throw std::runtime_error(ss.str());
		}
		/*
		std::cout << "host_particles\n";
		for (unsigned int i = 0; i < max; i++)
		{
			std::cout << h_particles[i].x<<", "<<h_particles[i].y<<"\n";
		}
		std::cout << "allParticles before\n";

		for (unsigned int i = 0; i < max; i++)
		{
			std::cout << (allParticles[i])->xy->x << ", " << allParticles[i]->xy->y << "\n";
		}
		*/

		
		for (unsigned int i = 0; i < max; i++)
		{
			*(allParticles[i])->xy = h_particles[i];
		}
		

		hipFree(d_force);
		hipFree(d_particles);
		delete[] h_particles;
		//delete[] h_force;

		/**/

		//do center last
		allParticles[0]->calcDistance(*forces1[0]);


		for (unsigned int i = 0; i < max; i++)
		{
			forces1[i]->reset();
		}
	

		/* end calc forces*/
		glfwSwapBuffers(window);
		glfwPollEvents();

		end = clock();
		time = (end - start);
		deltaTime += time;

		frames++;

		if (clockToMilliseconds(deltaTime) > 1000.0) { //every second
			frameRate = (double)frames*0.5 + frameRate * 0.5; //more stable
			frames = 0;
			deltaTime -= CLOCKS_PER_SEC;
			averageFrameTimeMilliseconds = 1000.0 / (frameRate == 0 ? 0.001 : frameRate);

			std::cout << "\tFrameTime was:\t[" << averageFrameTimeMilliseconds <<"]"<< std::endl;
			std::cout << "time per cycle: \t[" << time << "]" << std::endl;
		}

	}
	glfwTerminate();
}




int Galaxy::two_running_display(Galaxy& second)
{

	std::cout << "running display mode" << std::endl;
	Vector2D target;

	GLFWwindow* window;
	if (!glfwInit())
	{
		std::cout << "Error Initializing GLFW" << std::endl;
		return -1;
	}
	window = glfwCreateWindow(750, 750, "Barnes-Hut Tree: Two Galaxies", NULL, NULL);
	if (!window)
	{
		std::cout << "Error creating window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);

	std::vector<Vector2D> forces(NUMBER_PARTICLES);
	//size_t max = NUMBER_PARTICLES;

	clock_t deltaTime = 0;
	unsigned int frames = 0;


	std::clock_t end;
	std::clock_t start;
	double time;


	while (!glfwWindowShouldClose(window))
	{
		start = clock();

		//task parallel
		root->buildTree(allParticles, NUMBER_PARTICLES);

		//data parallel
		root->computeMassDistribution();

		displayParticles(allParticles, second.allParticles);
		//displayQuadrant(*root, *second.root);

		size_t max = allParticles.size();


		for (unsigned int i = 0; i < max; i++)
		{
			forces[i].reset();
		}

		//data parallel 

		tbb::parallel_for(size_t(1), max, [&](size_t i) 
		{
			root->calcForce(*(allParticles[i]),  (forces[i]));
			allParticles[i]->calcDistance(forces[i]);
		});

		//do center last
		root->calcForce(*(allParticles[0]),  (forces[0]));
		allParticles[0]->calcDistance(forces[0]);

		glfwSwapBuffers(window);
		glfwPollEvents();

		end = clock();;
		time = (end - start);
		deltaTime += time;

		frames++;

		if ( (deltaTime) >= 1.0) { //every second
			std::cout << 1000.0 / double(frames) << std::endl;

			double fps = double(frames) / deltaTime;

			std::cout << "\t fps was:\t[" << fps << "]" << std::endl;

			frames = 0;

			std::cout << "time per cycle: \t[" << time << "]" << std::endl;

		}

	}
	glfwTerminate();
}

/**/
double Galaxy::clockToMilliseconds(clock_t ticks) {
	return (ticks / (double)CLOCKS_PER_SEC)*1000.0;
}
/**/
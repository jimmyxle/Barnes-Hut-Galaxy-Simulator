


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void AddIntsCUDA(int* a, int *b)
{
	a[0] += b[0];
		
}

int main()
{
	//host
	int a = 5, b = 9;
	//device
	int *d_a, *d_b;

	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);


	AddIntsCUDA<<<1,1>>>(d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
	cout << "the answer is "<<a<<endl;

	hipFree(d_a);
	hipFree(d_b);

	int c;
	cin >> c;

	return 0;

}

/*
How to link directories and stuff 
go to youtube link:
https://www.youtube.com/watch?v=m0nhePeHwFs&list=PLKK11Ligqititws0ZOoGk3SW-TZCar4dK&index=1


keep this handy:
C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v8.0
to easily find incl and lib (use win32)

remember to get build dependencies and add CUDA 5.0
go to file and change item type to cuda/c++ (make sure to compile and make .exe file)




*/